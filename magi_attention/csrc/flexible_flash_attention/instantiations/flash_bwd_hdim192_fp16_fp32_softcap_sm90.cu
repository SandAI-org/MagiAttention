// Copyright (c) 2024, Jay Shah, Ganesh Bikshandi, Ying Zhang, Vijay Thakkar, Pradeep Ramani, Tri Dao.
// Splitting the different template instantiations to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_bwd_launch_template.h"

#ifndef FLASHATTENTION_DISABLE_HDIM192
template void run_mha_bwd_<90, cutlass::half_t, float, 192, true>(Flash_bwd_params &params, hipStream_t stream);
#endif
